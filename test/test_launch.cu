#include "gtest/gtest.h"
#include "hemi/launch.h"

struct k1 {
	template <typename... Arguments>
	HEMI_DEV_CALLABLE_MEMBER void operator()(int *count, Arguments... args) {
		*count = sizeof...(args); 
	}
};

TEST(LaunchTest, CorrectVariadicParams) {
	int *dCount;
	int count;
	hipMalloc(&dCount, sizeof(int));

	k1 kern;
	hemi::launch(kern, dCount, 1);
	hipMemcpy(&count, dCount, sizeof(int), hipMemcpyDefault);
	ASSERT_EQ(count, 1);

	hemi::launch(kern, dCount, 1, 2);
	hipMemcpy(&count, dCount, sizeof(int), hipMemcpyDefault);
	ASSERT_EQ(count, 2);

	hemi::launch(kern, dCount, 1, 2, 'a', 4.0, "hello");
	hipMemcpy(&count, dCount, sizeof(int), hipMemcpyDefault);
	ASSERT_EQ(count, 5);
	
}

struct k2 {
	template <typename... Arguments>
	HEMI_DEV_CALLABLE_MEMBER void operator()(int *bdim, int *gdim, Arguments... args) {
		*bdim = blockDim.x;
		*gdim = gridDim.x;
	}
};


TEST(LaunchTest, AutoConfigMaximalLaunch) {
	int *dBdim, *dGdim;
	int bdim, gdim;
	hipMalloc(&dBdim, sizeof(int));
	hipMalloc(&dGdim, sizeof(int));

	k2 kern;
	hemi::launch(kern, dBdim, dGdim);
	hipMemcpy(&bdim, dBdim, sizeof(int), hipMemcpyDefault);
	hipMemcpy(&gdim, dGdim, sizeof(int), hipMemcpyDefault);

	int devId;
	hipGetDevice(&devId);
	int smCount;
	hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, devId);
	ASSERT_GE(gdim, smCount);
	ASSERT_EQ(gdim%smCount, 0);
	ASSERT_GE(bdim, 1);
}
